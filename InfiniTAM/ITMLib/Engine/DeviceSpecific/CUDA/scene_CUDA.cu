#include "hip/hip_runtime.h"
//
// Created by Haoguang Huang on 18-5-14.
//

#include "scene_CUDA.h"
#include "hip/hip_runtime.h"
#include <>

#include <iostream>


//__global__ void process_kernel(test_struct* dev_ptr, const int N){
//    int tid = threadIdx.x + blockIdx.x * blockDim.x;
//
//    dev_ptr[tid].weight = tid;
//    dev_ptr[tid].sdf = tid * 2;
//}
//
//
//test_class::test_class() {
//    host_ptr = new test_struct[N];
//    hipMalloc((void**)&dev_ptr, sizeof(test_struct) * N);
//}
//
//
//test_class::~test_class() {
//    delete this->host_ptr;
//    hipFree(this->dev_ptr);
//}
//
//
/////__host__
//void test_class::process() {
//    hipError_t cudaStatus;
//    cudaStatus = hipMemcpy(dev_ptr, host_ptr, sizeof(test_struct)*N, hipMemcpyHostToDevice);
//    if(cudaStatus != hipSuccess){
//        fprintf(stderr, "hipMemcpyHostToDevice failed!");
//    }
//
//    //kernel
//    dim3 gridSize(1,1,1);
//    dim3 blockSize(16,1,1);
//    process_kernel<<<gridSize, blockSize>>>(dev_ptr, N);
//
//    cudaStatus = hipMemcpy(host_ptr, dev_ptr, sizeof(test_struct)*N, hipMemcpyDeviceToHost);
//    if(cudaStatus != hipSuccess){
//        fprintf(stderr, "hipMemcpyDeviceToHost failed!");
//    }
//
//    //output host_ptr
//    print();
//}
//
//
//void test_class::print() {
//    for(int i = 0; i < N; i++){
//        std::cout<<host_ptr[i].sdf<<"  "<<host_ptr[i].weight<<std::endl;
//    }
//}

